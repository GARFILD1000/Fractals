#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <malloc.h>

#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>

#define WIDTH 1280
#define HEIGHT 1024
#define VOLUME 3932160
#define ITERATIONS 24000

#define COLWIDTH 666
#define COLOFFSET 3200
#define BLACK 0

#define QCENTER 0.131825890901
#define PCENTER -0.743643900055

typedef thrust::tuple<char, char, char, int> tup;

//Хидер bmp файла
char head[]={0x42, 0x4D, 0x36, 0x00, 0x3C, 0x00, 0x00, 0x00, 0x00, 0x00, 0x36, 0x00, 0x00, 0x00, 0x28, 0x00, 0x00, 0x00, 0x00, 0x05, 0x00, 0x00, 0x00, 0x04, 0x00, 0x00, 0x01, 0x00, 0x18, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x3C, 0x00, 0x13, 0x0B, 0x00, 0x00, 0x13, 0x0B, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00};

struct speed_functor{
    double range;
    speed_functor(double _range):range(_range){}
    __host__ __device__ tup operator()(tup point)
    {
        int tupleIndex = thrust::get<3>(point);
        char col1 = 0, col2 = 0, col3 = 0;

        double Pmax = PCENTER + range/2;
        double Pmin = PCENTER - range/2;
        double Qmin = QCENTER - range/2.5;
        double dp = range/(WIDTH-1);
        double q = Qmin + tupleIndex/WIDTH * dp;
        double p = Pmin + tupleIndex%WIDTH * dp;
        double x, y, x2, y2, xtemp;
        int n;
        double cardio=sqrt((double)((p-0.25)*(p-0.25)+q*q));
        if (p<(cardio-2*cardio*cardio+0.25) || (p+1)*(p+1)+q*q<0.0625){
            col1 = BLACK;
            col2 = BLACK;
            col3 = BLACK;
        }
        else{
            n = x = y = x2 = y2 = 0;
            while(n < ITERATIONS){
            if (x2 + y2 > 4)
                break;
                xtemp=x2-y2+p;
                y=2*x*y+q;
                x=xtemp;
                x2=x*x;
                y2=y*y;
                n++;
            }
            if (n==ITERATIONS) 
            {
              col1 = BLACK;
              col2 = BLACK;
              col3 = BLACK;
            }
            else 
            { 
              n=(n+COLOFFSET)%(3*COLWIDTH);
              if (n/COLWIDTH==0)
              { // Голубой 116.11.0          
                col1 = 116+139*n/COLWIDTH;
                col2 = 11+244*n/COLWIDTH;
                col3 = 237*n/COLWIDTH;
              }
              else if (n/COLWIDTH==1)
              { // Белый 255.255.237
                n-=COLWIDTH;
                col1 = 255-253*n/COLWIDTH;
                col2 = 255-123*n/COLWIDTH;
                col3 = 238;
              }
              else 
              { // Рыжий 2.132.239
                n-=2*COLWIDTH;
                col1 = 2+114*n/COLWIDTH;
                col2 = 132-121*n/COLWIDTH;
                col3 = 239-239*n/COLWIDTH;
              }
            }
        }
        return thrust::make_tuple(col1, col2, col3, tupleIndex);
    }
};

bool fileSaving = false;


void saveData(char fileName[],char *results){
    printf("Creating file %s\n", fileName);
    FILE *file = fopen(fileName, "w");
    
    if(file == NULL){
        return;
    }
    fwrite(head, 1, 54, file);
    fwrite(results, 1, VOLUME, file);
    fclose(file);
}

__global__ void setSpeed(char *results, double range){
    int deviceIndex = threadIdx.x + blockDim.x * blockIdx.x;
    double Pmax = PCENTER + range/2;
    double Pmin = PCENTER - range/2;
    double Qmin = QCENTER - range/2.5;
    double dp = range/(WIDTH-1);
    double q = Qmin + deviceIndex/WIDTH * dp;
    double p = Pmin + deviceIndex%WIDTH * dp;
    double x, y, x2, y2, xtemp;
    int n;
    double cardio=sqrt((double)((p-0.25)*(p-0.25)+q*q));
    if (p<(cardio-2*cardio*cardio+0.25) || (p+1)*(p+1)+q*q<0.0625){
        results[deviceIndex*3] = BLACK;
        results[deviceIndex*3+1] = BLACK;
        results[deviceIndex*3+2] = BLACK;
    }
    else{
        n = x = y = x2 = y2 = 0;
        while(n < ITERATIONS){
        if (x2 + y2 > 4)
            break;
            xtemp=x2-y2+p;
            y=2*x*y+q;
            x=xtemp;
            x2=x*x;
            y2=y*y;
            n++;
        }
        if (n==ITERATIONS) 
        {
          results[deviceIndex*3] = BLACK;
          results[deviceIndex*3+1] = BLACK;
          results[deviceIndex*3+2] = BLACK;
        }
        else 
        { 
          n=(n+COLOFFSET)%(3*COLWIDTH);
          if (n/COLWIDTH==0)
          { // Голубой 116.11.0          
            results[deviceIndex*3] = 116+139*n/COLWIDTH;
            results[deviceIndex*3+1] = 11+244*n/COLWIDTH;
            results[deviceIndex*3+2] = 237*n/COLWIDTH;
          }
          else if (n/COLWIDTH==1)
          { // Белый 255.255.237
            n-=COLWIDTH;
            results[deviceIndex*3] = 255-253*n/COLWIDTH;
            results[deviceIndex*3+1] = 255-123*n/COLWIDTH;
            results[deviceIndex*3+2] = 238;
          }
          else 
          { // Рыжий 2.132.239
            n-=2*COLWIDTH;
            results[deviceIndex*3] = 2+114*n/COLWIDTH;
            results[deviceIndex*3+1] = 132-121*n/COLWIDTH;
            results[deviceIndex*3+2] = 239-239*n/COLWIDTH;
          }
        }
    }
    return;
}


double pureCUDA(double range)
{
	hipEvent_t start, stop;
	float elapsedTime, allTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
    char *results_h;
    char *results_d;
    
    int setSize = 0;
    
    printf("Start pureCUDA fractal calculating...\n");
    
    hipMalloc((void**)&results_d, sizeof(char)*(VOLUME));
    hipHostMalloc((void**)&results_h, sizeof(char)*(VOLUME));
    
    int threadsPerBlock = 32;
    int numOfBlocks = WIDTH * HEIGHT / threadsPerBlock;
    
    while(range > 0.00000005){
        setSize++;
        hipEventRecord(start, 0);
        setSpeed <<< dim3(numOfBlocks), dim3(threadsPerBlock) >>> (results_d, range);
        hipDeviceSynchronize();
        
        
        hipMemcpy(results_h, results_d, (VOLUME)*sizeof(char), hipMemcpyDeviceToHost);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
	    hipEventElapsedTime(&elapsedTime, start, stop);
        
        printf("Fractal calculated! Time = %f ms.\n", elapsedTime);
        char numberInString[100];
        snprintf(numberInString, 100, "pureCUDA/Output%d", setSize);
        if (fileSaving) saveData(numberInString,results_h);
        allTime += elapsedTime;
        range /= 2;
    }
    
    
    hipFree(results_d);
    hipFree(results_h);
    return allTime/setSize;
}


double thrustCUDA(double range)
{
	hipEvent_t start, stop;
	float elapsedTime, allTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	thrust::host_vector<char> color1_h(WIDTH*HEIGHT);
	thrust::host_vector<char> color2_h(WIDTH*HEIGHT);
	thrust::host_vector<char> color3_h(WIDTH*HEIGHT);
	thrust::host_vector<int> index_h(WIDTH*HEIGHT);
	
	thrust::device_vector<char> color1_d(WIDTH*HEIGHT);
	thrust::device_vector<char> color2_d(WIDTH*HEIGHT);
	thrust::device_vector<char> color3_d(WIDTH*HEIGHT);
	thrust::device_vector<int> index_d(WIDTH*HEIGHT);
	
	thrust::fill(color1_d.begin(), color1_d.end(), 0);
	thrust::fill(color2_d.begin(), color2_d.end(), 0);
	thrust::fill(color3_d.begin(), color3_d.end(), 0);
	thrust::sequence(index_d.begin(), index_d.end());
	
    char *results = (char*)calloc(sizeof(char), VOLUME);
    int setSize = 0;
    printf("Start thrustCUDA fractal calculating...\n");
     
    while(range > 0.00000005){
    speed_functor speedFunctor(range);
        setSize++;
        hipEventRecord(start, 0);
        thrust::transform(
            thrust::make_zip_iterator(
                thrust::make_tuple(color1_d.begin(), color2_d.begin(), color3_d.begin(), index_d.begin())
            ),
            thrust::make_zip_iterator(
                thrust::make_tuple(color1_d.end(), color2_d.end(), color3_d.end(),
                    index_d.end())
            ),
            thrust::make_zip_iterator(
                thrust::make_tuple(color1_d.begin(), color2_d.begin(), color3_d.begin(), index_d.begin())
            ),
            speedFunctor
        );
        
        //hipDeviceSynchronize();
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
	    hipEventElapsedTime(&elapsedTime, start, stop);
	    
        //results_h = results_d;
        
        color1_h = color1_d;
        color2_h = color2_d;
        color3_h = color3_d;
        for(int i = 0; i < WIDTH*HEIGHT; i++){
            results[i*3] = color1_h[i];
            results[i*3+1] = color2_h[i];
            results[i*3+2] = color3_h[i];
        }
        
        printf("Fractal calculated! Time = %f ms.\n", elapsedTime);
        char numberInString[100];
        snprintf(numberInString, 100, "thrustCUDA/Output%d", setSize);
        if (fileSaving) saveData(numberInString,results);
        range /= 2;
        
        allTime+= elapsedTime;
    }
    
    
    //hipFree(results_d);
    //hipFree(results_h);
    return allTime/setSize;
}

int main()
{
    printf("Start comparing program\n");
    fileSaving = false;
    double range = 0.000005;
    double pureCudaTime = 0;
    double thrustCudaTime = 0;
    
    thrustCudaTime = thrustCUDA(range);
    pureCudaTime = pureCUDA(range);
    
    printf("Time:\nPure CUDA:%f\nThrust:%f\n", pureCudaTime, thrustCudaTime);
    return 0;
}

